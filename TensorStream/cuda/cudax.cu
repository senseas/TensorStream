
#include <hip/hip_runtime.h>

#include <stdio.h>

template<typename T>
__global__ void addKernel(T* c, const T* a, const T* b) {
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}

template<typename T>
__global__ void sigmoidKernel(T* inx, T* out) {
  int i = threadIdx.x;
  out[i] = 1 / (1 + exp(inx[i]));
  printf("%f\n", out[0]);
}

template<typename T>
__global__ void matmulKernel(T* c, T* a, T* b, unsigned int h, unsigned int w, unsigned int n) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int idy = blockDim.y * blockIdx.y + threadIdx.y;

  //printf("x={%i}\n", idx);
  //printf("y={%i}\n", idx);

  T sum = 0;
  for (size_t i = 0; i < w; i++){
	  sum += a[idy * w + i] * b[i * n + idx];
  }

  c[idy * n + idx] = sum;
}
