#pragma once
#include <hip/hip_runtime.h>


#include <stdio.h>

template<typename T>
__global__ void addKernel(T* c, const T* a, const T* b) {
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}

template<typename T>
__global__ void sigmoidKernel(T* inx, T* out) {
  int i = threadIdx.x;
  out[i] = 1 / (1 + exp(inx[i]));
  //printf("%f\n", out[0]);
}

template<typename T>
__global__ void matmulKernel(T* a, T* b, T* c, int h, int w, int n) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int idy = blockDim.y * blockIdx.y + threadIdx.y;

  //printf("x={%i}\n", idx);
  //printf("y={%i}\n", idy);

  T sum = 0;
  for (size_t i = 0; i < n; i++) {
    sum += a[idx * n + i] * b[i * w + idy];
  }

  c[idy * n + idx] = sum;
}

template <typename Func> 
__global__ void _forEach(size_t N, Func func) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride = gridDim.x * blockDim.x;
  for (size_t i = id; i < N; i += stride) {
    func(i);
  }
}

template <typename Func>
__global__ void _forEach2(size_t M, size_t N, Func func) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int idy = blockDim.y * blockIdx.y + threadIdx.y;

  size_t stridex = gridDim.x * blockDim.x;
  size_t stridey = gridDim.y * blockDim.y;
  for (size_t x = idx; x < M; x += stridex) {
    for (size_t y = idy; y < N; y += stridey) {
      func(x, y);
    }
  }
}