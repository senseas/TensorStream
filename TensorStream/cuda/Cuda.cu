#pragma once
#include <hip/hip_runtime.h>


#include <stdio.h>

template <typename T> 
__global__ void addKernel(T *c, const T *a, const T *b) {
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}

template <typename T> 
__global__ void sigmoidKernel(T *inx, T *out) {
  int i = threadIdx.x;
  out[i] = 1 / (1 + exp(inx[i]));
  // printf("%f\n", out[0]);
}

template <typename T>
__global__ void matmulKernel(T *a, T *b, T *c, int h, int w, int n) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int idy = blockDim.y * blockIdx.y + threadIdx.y;

  // printf("x={%i}\n", idx);
  // printf("y={%i}\n", idy);

  T sum = 0;
  for (size_t i = 0; i < n; i++) {
    sum += a[idx * n + i] * b[i * w + idy];
  }

  c[idy * n + idx] = sum;
}

template <typename Func> 
__global__ void cudaForEach1D(size_t N, Func func) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (N); i += blockDim.x * gridDim.x) {
    func(i);
  }
}

template <typename Func>
__global__ void cudaForEach2D(size_t M, size_t N, Func func) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (M); i += blockDim.x * gridDim.x) {
    for (size_t j = blockIdx.y * blockDim.y + threadIdx.y; j < (N); j += blockDim.y * gridDim.y) {
      func(i, j);
    }
  }
}