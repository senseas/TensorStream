#pragma once
#include <hip/hip_runtime.h>


#include <stdio.h>

template<typename T>
__global__ void addKernel(T* c, const T* a, const T* b) {
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}

template<typename T>
__global__ void sigmoidKernel(T* inx, T* out) {
  int i = threadIdx.x;
  out[i] = 1 / (1 + exp(inx[i]));
  //printf("%f\n", out[0]);
}

template<typename T>
__global__ void matmulKernel(T* a, T* b, T* c, int h, int w, int n) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int idy = blockDim.y * blockIdx.y + threadIdx.y;

  //printf("x={%i}\n", idx);
  //printf("y={%i}\n", idy);

  T sum = 0;
  for (size_t i = 0; i < n; i++) {
    sum += a[idx * n + i] * b[i * w + idy];
  }

  c[idy * n + idx] = sum;
}
